#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <random>
#include <omp.h>
#include <omp.h>
#include <climits> 
#include <math.h>
#include <vector>
using namespace std;

#define bs 256
#define SEED 123
#define DIGITS 10

// Cada thread toma el digito de un elemento del array A con respecto al exponente dado y aumenta
// un contador de ocurrencias de dicho digito
__global__ void histogram_kernel(int n, int *dA, int *dHist, int exp) {
    // Histograma con ocurrencias de cada digito en memoria compartida
    __shared__ int sharedHist[DIGITS];
    
    // Se limpia el histograma de resultados anteriores
    if (threadIdx.x < DIGITS) sharedHist[threadIdx.x] = 0;
    __syncthreads();

    // Se obtiene digito de elemento de A y se usa atomicAdd para añadir ocurrencia a sharedHist
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tidx < n) {
        int digit = (dA[tidx] / exp) % DIGITS;
        atomicAdd(&sharedHist[digit], 1);
    }
    __syncthreads();

    // Se guarda resultados de memoria compartida en el histograma dHist
    if (threadIdx.x < DIGITS) {
        atomicAdd(&dHist[threadIdx.x], sharedHist[threadIdx.x]);
    }
}

// 
__global__ void scatter_kernel(int n, int *dA, int *dR, int *dPrefixSum, int exp) {
    // prefixSum en memoria compartida
    __shared__ int sharedPrefix[DIGITS];

    // se carga la suma de prefijos a la memoria compartida para mejorar rendimiento 
    if (threadIdx.x < DIGITS) {
        sharedPrefix[threadIdx.x] = dPrefixSum[threadIdx.x];
    }
    __syncthreads();

    // Se obtiene digito del elemento de A al que accede el thread
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tidx < n) {
        int digit = (dA[tidx] / exp) % DIGITS;

        // Se obtiene posicion global que ocupara el elemento de A y se guarda en R
        int pos = atomicAdd(&sharedPrefix[digit], 1);
        dR[pos] = dA[tidx];
    }
}

// Realiza prefixSum con el histograma y versiones anteriores del prefixSum y guarda en el mismo prefix
void exclusive_scan(int *hist, int *prefix, int size) {
    prefix[0] = 0;
    for (int i = 1; i < size; ++i) {
        prefix[i] = prefix[i - 1] + hist[i - 1];
    }
}

// PSKC Radix-sort
void cpu(int *&data, int n, int b);

void gpu(int *A, int *R, int *dA, int *dR, int n);

// llena array de tamaño n con enteros
void llena_array(int *A, int n, int nt, int seed);

// Imprime un array de enteros
void print_array(int n, int *array);

// Ejecutar como ./prog n modo nt b
int main(int argc, char **argv){

    // Tomar argumentos e inicializar variables
    int n = atoi(argv[1]);
    int modo = atoi(argv[2]); 
    int nt = atoi(argv[3]);
    int b = atoi(argv[4]);
    omp_set_num_threads(nt);

    int *A = new int[n];
    int *R = new int[n]; // arreglos en memoria principal
    int *dA;
    int *dR; // direcciones de arreglos en gpu

    // Llenar array e imprimir si es lo bastante pequeño
    llena_array(A, n, nt, SEED);
    if(n <= 32){
        cout << "Array de entrada:" << endl;
        print_array(n, A);
    }

    if (modo == 0){
        double inicio = omp_get_wtime();
        cpu(R, n, b);
        double fin = omp_get_wtime();
        cout << "Tiempo de ejecución: " << fin - inicio << " segundos" << endl;
    }
    else{
        // allocar memoria en device  (GPU)
        hipMalloc(&dA, n * sizeof(int));
        hipMalloc(&dR, n * sizeof(int));

        // copiar de Host -> Device
        hipMemcpy(dA, A, sizeof(int)*n, hipMemcpyHostToDevice);
        hipMemcpy(dR, R, sizeof(int)*n, hipMemcpyHostToDevice);

        gpu(A, R, dA, dR, n);
    }

    // Imprimir resultado si es lo bastante pequeño
    if(n<= 32){
        cout << "Array de salida:" << endl;
        print_array(n, R);
    }

    // Liberar memoria
    delete[] A;
    delete[] R;
}

void llena_array(int *A, int n, int nt, int seed){
    #pragma omp parallel shared(A)
    {
        int tid = omp_get_thread_num();
        long chunk = n / nt;
        long start = tid * chunk;
        long end = (tid == nt - 1) ? n : start + chunk;

        std::mt19937 mt(seed + tid);
        std::uniform_int_distribution<int> dist(0, n - 1);

        for (int k = start; k < end; ++k) {
            A[k] = dist(mt);
        }
    }
}

void print_array(int n, int *array){
    for(int i = 0; i < n; ++i){
        printf("%d ", array[i]);
    }
    cout << endl;
}

void cpu(int *&data, int n, int b){
    cout << "Comenzando..en funcion" << endl;
    // Paso 1: Obtener muestra de datos
    // Sample: contiene elementos seleccionados a intervalos regulares del vector data, con el objetivo no tener que procesar todo el vector original
    int* sample = new int[n];
    #pragma omp parallel for
    for (int i = 0; i < n; i++) {
        sample[i] = data[i * (n / b)];
    }
    cout << "listo paso 1" << endl;
    // Paso 2: Reverse Sorting local
    // El Reverse Sorting local se refiere a ordenar un subconjunto (sample) del vector data en orden descendente (de mayor a menor).
    sort(sample, sample + n); // ordena el array sample de manera ascendente (de menor a mayor)
    
    // A partir de los valores ordenados en sample, se seleccionan b valores que servirán como particiones. Estos valores se almacenan en el arreglo partitions.

    vector<int> partitions(b + 1); // b particiones
    // se están eligiendo b elementos equidistantes del vector sample ordenado, que servirán como valores de partición.
    for (int i = 0; i < b; i++) {
        partitions[i] = sample[i * (n / b)]; 
    }
    partitions[b] = INT_MAX; // para tener un valor de partición que sea mayor que cualquier elemento del vector data.
    cout << "listo paso 2" << endl;
    // Paso 3: Comprobar equilibrio de carga y elección
    
    // Se averigua si se podria conseguir un buen equilibrio de carga entre los procesadores tras dos iteraciones de Reverse Sorting. 

    int total_work = 0;
    #pragma omp parallel for reduction(+:total_work)
    for (int i = 0; i < n; i++) {
        int bucket = std::lower_bound(partitions.begin(), partitions.end(), data[i]) - partitions.begin() - 1;
        total_work += bucket;
    }
    
    double imbalance = static_cast<double>(total_work) / (n * b);
    cout << "listo paso 3" << endl;
    // Si aplicando Reverse Sorting NO se consiguiese un buen equilibrio de carga entre los datos muestreados, se estara en el Caso (3.2) 
    if (imbalance > 1.1) {
        // Caso 3.1: Aplicar Reverse Sorting paralelo y ordenar.
        // Todos los procesadores particionan el conjunto de datos a ordenar con la tecnica de Reverse Sorting paralelo.
        cout << "en caso 3.1" << endl;
        #pragma omp parallel for
        for (int i = 0; i < b; i++) {
            sort(data + i * (n / b), data + (i + 1) * (n / b), greater<int>());
        }
    } else {
        // Caso 3.2: Aplicar Counting Split paralelo y ordenar.
        // Los procesadores particionan conCounting Split en paralelo. 
        cout << "en caso 3.2" << endl;
        vector<vector<int>> buckets(b);
        #pragma omp parallel for
        for (int i = 0; i < n; i++) {
            int bucket_index = lower_bound(partitions.begin(), partitions.end(), data[i]) - partitions.begin() - 1;
            #pragma omp critical
            {
                buckets[bucket_index].push_back(data[i]);
            }
        }

        #pragma omp parallel for
        for (int i = 0; i < b; i++) {
            sort(buckets[i].begin(), buckets[i].end());
        }

        int index = 0;
        for (const auto& bucket : buckets) {
            for (int num : bucket) {
                data[index++] = num;
            }
        }
    }

    // Limpiar memoria de particiones y muestra
    delete[] sample;
}


void gpu(int *A, int *R, int *dA, int *dR, int n) {
    // Reservar memoria en gpu para histograma y prefixsum
    int *dHist, *dPrefixSum;
    hipMalloc(&dHist, DIGITS * sizeof(int));
    hipMalloc(&dPrefixSum, DIGITS * sizeof(int));

    // Dimensiones de la grilla
    dim3 blockSize(bs, 1, 1);
    dim3 gridSize((n + bs - 1) / bs, 1, 1);

    // Inicializar histograma y prefixsum en memoria principal
    int *hHist = new int[DIGITS];
    int *hPrefixSum = new int[DIGITS];

    // Para medir tiempo en milisegundos
    hipEvent_t start, stop;
    float milliseconds = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Comienza a medir tiempo
    hipEventRecord(start);

    // A partir del numero mas grande del array se sabe cuantos digitos usar
    int maxVal = *max_element(A, A + n);

    // El for se ejecuta por cada digito, es decir unidades, decenas, etc
    for (int exp = 1; maxVal / exp > 0; exp *= 10) {
        // Se resetea el histograma de gpu
        hipMemset(dHist, 0, DIGITS * sizeof(int));

        // Se crea el histograma para los digitos actuales
        histogram_kernel<<<gridSize, blockSize>>>(n, dA, dHist, exp);
        hipDeviceSynchronize();

        // Se copia el histograma de gpu al de memoria principal
        hipMemcpy(hHist, dHist, DIGITS * sizeof(int), hipMemcpyDeviceToHost);

        // Se calcula prefixSum
        exclusive_scan(hHist, hPrefixSum, DIGITS);

        // Se copia prefixSum en memoria principal
        hipMemcpy(dPrefixSum, hPrefixSum, DIGITS * sizeof(int), hipMemcpyHostToDevice);

        // Se ordenan elementos de acuerdo al digito actual con el prefixSum 
        scatter_kernel<<<gridSize, blockSize>>>(n, dA, dR, dPrefixSum, exp);
        hipDeviceSynchronize();

        // Se intercambian punteros de A y R para trabajar sobre el arreglo parcialmente ordenado
        // con respecto al ultimo digito
        std::swap(dA, dR);
    }

    // Se termina de contar el tiempo
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Se guarda el resultado en memoria principal
    hipMemcpy(R, dA, n * sizeof(int), hipMemcpyDeviceToHost);

    // Se imprime tiempo que tomo ordenar
    std::cout << "Time taken by GPU radix sort: " << milliseconds << " ms" << std::endl;

    // Liberar memoria
    hipFree(dHist);
    hipFree(dPrefixSum);
    delete[] hHist;
    delete[] hPrefixSum;

    // Se eliminan eventos de cuda
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

