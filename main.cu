#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>
using namespace std;

#define bs 256


#define DIGITS 10

__global__ void histogram_kernel(int n, int *dA, int *dHist, int exp) {
    __shared__ int sharedHist[DIGITS];
    
    // Initialize shared histogram
    if (threadIdx.x < DIGITS) sharedHist[threadIdx.x] = 0;
    __syncthreads();

    // Compute digit for each element
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tidx < n) {
        int digit = (dA[tidx] / exp) % DIGITS;
        atomicAdd(&sharedHist[digit], 1);
    }
    __syncthreads();

    // Write shared histogram to global memory
    if (threadIdx.x < DIGITS) {
        atomicAdd(&dHist[threadIdx.x], sharedHist[threadIdx.x]);
    }
}

__global__ void scatter_kernel(int n, int *dA, int *dR, int *dPrefixSum, int exp) {
    __shared__ int sharedPrefix[DIGITS];

    // Load prefix sums into shared memory
    if (threadIdx.x < DIGITS) {
        sharedPrefix[threadIdx.x] = dPrefixSum[threadIdx.x];
    }
    __syncthreads();

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tidx < n) {
        int digit = (dA[tidx] / exp) % DIGITS;

        // Compute global position and scatter element
        int pos = atomicAdd(&sharedPrefix[digit], 1);
        dR[pos] = dA[tidx];
    }
}

void exclusive_scan(int *hist, int *prefix, int size) {
    prefix[0] = 0;
    for (int i = 1; i < size; ++i) {
        prefix[i] = prefix[i - 1] + hist[i - 1];
    }
}


void cpu();

void gpu(int *A, int *R, int *dA, int *dR, int n);

// llena array de tamaño n con enteros
void llena_array(int n, int *array);

// Imprime un array de enteros
void print_array(int n, int *array);

// Ejecutar como ./prog n modo nt
int main(int argc, char **argv){
    srand(static_cast<unsigned int>(time(0)));

    // Tomar argumentos e inicializar variables
    int n = atoi(argv[1]);
    int modo = atoi(argv[2]); 
    int nt = atoi(argv[3]);

    int *A = new int[n];
    int *R = new int[n]; // arreglos en memoria principal
    int *dA;
    int *dR; // direcciones de arreglos en gpu
    
    // Llenar array e imprimir si es lo bastante pequeño
    llena_array(n, A);
    if(n <= 32){
        cout << "Array de entrada:" << endl;
        print_array(n, A);
    }

    if (modo == 0){
        cpu();
    }
    else{
        // allocar memoria en device  (GPU)
        hipMalloc(&dA, n * sizeof(int));
        hipMalloc(&dR, n * sizeof(int));

        // copiar de Host -> Device
        hipMemcpy(dA, A, sizeof(int)*n, hipMemcpyHostToDevice);
        hipMemcpy(dR, R, sizeof(int)*n, hipMemcpyHostToDevice);

        gpu(A, R, dA, dR, n);


    }

    // Imprimir resultado si es lo bastante pequeño
    if(n<= 32){
        cout << "Array de salida:" << endl;
        print_array(n, R);
    }

    // Liberar memoria
    delete[] A;
    delete[] R;

}

void llena_array(int n, int *array) {
    for (int i = 0; i < n; ++i) {
        array[i] = rand() % 1000;
    }
}

void print_array(int n, int *array){
    for(int i = 0; i < n; ++i){
        printf("%d ", array[i]);
    }
    cout << endl;
}

void cpu(){
    return;
}

void gpu(int *A, int *R, int *dA, int *dR, int n) {
    // Allocate memory for histogram and prefix sum
    int *dHist, *dPrefixSum;
    hipMalloc(&dHist, DIGITS * sizeof(int));
    hipMalloc(&dPrefixSum, DIGITS * sizeof(int));

    dim3 blockSize(bs, 1, 1);
    dim3 gridSize((n + bs - 1) / bs, 1, 1);

    int *hHist = new int[DIGITS];
    int *hPrefixSum = new int[DIGITS];

    // Create CUDA events for timing
    hipEvent_t start, stop;
    float milliseconds = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the timer
    hipEventRecord(start);

    int maxVal = *max_element(A, A + n);
    for (int exp = 1; maxVal / exp > 0; exp *= 10) {
        // Reset histogram
        hipMemset(dHist, 0, DIGITS * sizeof(int));

        // Step 1: Compute histogram
        histogram_kernel<<<gridSize, blockSize>>>(n, dA, dHist, exp);
        hipDeviceSynchronize();

        // Copy histogram to host and compute prefix sum
        hipMemcpy(hHist, dHist, DIGITS * sizeof(int), hipMemcpyDeviceToHost);
        exclusive_scan(hHist, hPrefixSum, DIGITS);

        // Copy prefix sum to device
        hipMemcpy(dPrefixSum, hPrefixSum, DIGITS * sizeof(int), hipMemcpyHostToDevice);

        // Step 2: Scatter elements
        scatter_kernel<<<gridSize, blockSize>>>(n, dA, dR, dPrefixSum, exp);
        hipDeviceSynchronize();

        // Swap input and output arrays for next iteration
        std::swap(dA, dR);
    }

    // End the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy final sorted array to host
    hipMemcpy(R, dA, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the time taken
    std::cout << "Time taken by GPU radix sort: " << milliseconds << " ms" << std::endl;

    // Free memory
    hipFree(dHist);
    hipFree(dPrefixSum);
    delete[] hHist;
    delete[] hPrefixSum;

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

